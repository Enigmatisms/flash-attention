#include "flash_bwd_launch_template.h"

#ifndef FLASHATTENTION_DISABLE_HDIM192
template<>
void run_mla_bwd_<90, cutlass::bfloat16_t, 192, false>(Flash_bwd_params &params, hipStream_t stream) {
  printf("\nwsm debug run_mla_bwd_()\n");
  run_mha_bwd_hdim192_hdimv128<90, cutlass::bfloat16_t, false>(params, stream);
}
template<>
void run_mla_bwd_<90, cutlass::bfloat16_t, 192, true>(Flash_bwd_params &params, hipStream_t stream) {
  printf("\nwsm debug run_mla_bwd_()\n");
  run_mha_bwd_hdim192_hdimv128<90, cutlass::bfloat16_t, true>(params, stream);
}
#endif
