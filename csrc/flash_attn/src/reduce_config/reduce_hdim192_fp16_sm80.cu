#include "reducelaunch_template.h"

template<>
void run_reduce_<cutlass::half_t, 192>(Reduce_attn_scores_params &params, hipStream_t stream) {
    run_reduce_hdim192<cutlass::half_t>(params, stream);
}
