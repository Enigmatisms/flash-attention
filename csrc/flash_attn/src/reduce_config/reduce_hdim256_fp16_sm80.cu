#include "reduce_launch_template.h"

template<>
void run_reduce_<cutlass::half_t, 256>(Reduce_attn_scores_params &params, hipStream_t stream) {
    run_reduce_hdim256<cutlass::half_t>(params, stream);
}
